#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <math.h>
#include <chrono>
#include <fstream>
#include <iomanip>

// ====== OpenGL / CUDA-OpenGL Interop ======
#include <GL/glew.h>       // if using GLEW
#include <GL/freeglut.h>   // if using freeGLUT
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <string>
#include <sstream>

// Time info
int currentStep = 0;
std::chrono::time_point<std::chrono::high_resolution_clock> startTime;
float updatesPerSecond = 0.0f;
float mlups = 0.0f;


// Force the use of the NVIDIA GPU
#ifdef _WIN32
extern "C" {
    __declspec(dllexport) DWORD NvOptimusEnablement = 0x00000001;
}
#endif

//-----------------------------------------------------
// Lattice parameters and simulation constants
//-----------------------------------------------------
const int nx = 512;
const int ny = 512;
const int numDirs = 9;
typedef float DTYPE;

__device__ int cx_const[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
__device__ int cy_const[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
__device__ DTYPE w_const[9] = {
    4.0f / 9.0f,
    1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f,
    1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
};

int cx[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
int cy[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
DTYPE w[9] = {
    4.0f / 9.0f,
    1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f, 1.0f / 9.0f,
    1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
};

DTYPE U = 0.4f;
DTYPE Re = 7000.0f;
DTYPE nu, omega;  // nu = 3*(U*nx/Re)+0.5; omega = 1/nu

// Simulation arrays (device pointers)
DTYPE* d_f = nullptr, * d_f_new = nullptr;
char* d_mask = nullptr;

size_t simSize = nx * ny * numDirs * sizeof(DTYPE);
size_t maskSize = nx * ny * sizeof(char);

//-----------------------------------------------------
// Helper device and host inline for indexing
//-----------------------------------------------------
__device__ inline int idx(int i, int j, int k, int nx, int ny) {
    return i + j * nx + k * nx * ny;
}
inline int idx_h(int i, int j, int k, int nx, int ny) {
    return i + j * nx + k * nx * ny;
}

//-----------------------------------------------------
// CUDA kernels for the LBM solver
//-----------------------------------------------------
__global__ void collision_kernel(DTYPE* f, DTYPE omega, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        DTYPE rho = 0.0f;
        DTYPE u_x = 0.0f;
        DTYPE u_y = 0.0f;
        for (int k = 0; k < numDirs; k++) {
            DTYPE val = f[idx(i, j, k, nx, ny)];
            rho += val;
            u_x += val * cx_const[k];
            u_y += val * cy_const[k];
        }
        if (rho > 0.0f) {
            u_x /= rho;
            u_y /= rho;
        }
        DTYPE usqr = u_x * u_x + u_y * u_y;
        for (int k = 0; k < numDirs; k++) {
            DTYPE cu = 3.0f * (cx_const[k] * u_x + cy_const[k] * u_y);
            DTYPE feq = w_const[k] * rho * (1.0f + cu + 0.5f * cu * cu - 1.5f * usqr);
            f[idx(i, j, k, nx, ny)] = (1.0f - omega) * f[idx(i, j, k, nx, ny)] + omega * feq;
        }
    }
}

__global__ void streaming_kernel(DTYPE* f_in, DTYPE* f_out, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
            for (int k = 0; k < numDirs; k++) {
                int ip = i - cx_const[k];
                int jp = j - cy_const[k];
                f_out[idx(i, j, k, nx, ny)] = f_in[idx(ip, jp, k, nx, ny)];
            }
        }
        else {
            // For simplicity, no wrap-around. Just copy as-is at borders
            for (int k = 0; k < numDirs; k++) {
                f_out[idx(i, j, k, nx, ny)] = f_in[idx(i, j, k, nx, ny)];
            }
        }
    }
}

__global__ void bounce_back_kernel(DTYPE* f, char* mask, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        if (mask[i + j * nx] == 1) {
            // Swap east (1) and west (3)
            int idx1 = idx(i, j, 1, nx, ny);
            int idx3 = idx(i, j, 3, nx, ny);
            DTYPE tmp = f[idx1];
            f[idx1] = f[idx3];
            f[idx3] = tmp;
            // Swap north (2) and south (4)
            int idx2 = idx(i, j, 2, nx, ny);
            int idx4 = idx(i, j, 4, nx, ny);
            tmp = f[idx2];
            f[idx2] = f[idx4];
            f[idx4] = tmp;
            // Swap NE (5) and SW (7)
            int idx5 = idx(i, j, 5, nx, ny);
            int idx7 = idx(i, j, 7, nx, ny);
            tmp = f[idx5];
            f[idx5] = f[idx7];
            f[idx7] = tmp;
            // Swap NW (6) and SE (8)
            int idx6 = idx(i, j, 6, nx, ny);
            int idx8 = idx(i, j, 8, nx, ny);
            tmp = f[idx6];
            f[idx6] = f[idx8];
            f[idx8] = tmp;
        }
    }
}

__global__ void moving_lid_kernel(DTYPE* f, int nx, int ny, DTYPE U) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nx) {
        int j = ny - 2;
        // Zou-He velocity BC
        //DTYPE rho = f[idx(i, j, 0, nx, ny)] + f[idx(i, j, 1, nx, ny)] + f[idx(i, j, 3, nx, ny)]
        //    + 2.0f * (f[idx(i, j, 2, nx, ny)] + f[idx(i, j, 5, nx, ny)] + f[idx(i, j, 6, nx, ny)]);
        //    f[idx(i, j, 4, nx, ny)] = f[idx(i, j, 2, nx, ny)];
        //    f[idx(i, j, 7, nx, ny)] = f[idx(i, j, 5, nx, ny)]
        //        + 0.5f * (f[idx(i, j, 1, nx, ny)] - f[idx(i, j, 3, nx, ny)])
        //            - 0.5f * rho * U;
        //        f[idx(i, j, 8, nx, ny)] = f[idx(i, j, 6, nx, ny)]
        //            - 0.5f * (f[idx(i, j, 1, nx, ny)] - f[idx(i, j, 3, nx, ny)])
        //                + 0.5f * rho * U;

		// Mid-grid velocity BC
		f[idx(i, j, 4, nx, ny)] = f[idx(i, j, 2, nx, ny)];
		f[idx(i, j, 7, nx, ny)] = f[idx(i, j, 5, nx, ny)] - DTYPE(1.0) / DTYPE(6.0) * U;
		f[idx(i, j, 8, nx, ny)] = f[idx(i, j, 6, nx, ny)] + DTYPE(1.0) / DTYPE(6.0) * U;
    }
}

//-----------------------------------------------------
// Kernel to compute velocity magnitude into a float array
//-----------------------------------------------------
__global__ void compute_velocity_field_kernel(const DTYPE* f, DTYPE* velocity_mag, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        DTYPE rho = 0.0f;
        DTYPE u_x = 0.0f;
        DTYPE u_y = 0.0f;
        for (int k = 0; k < numDirs; k++) {
            DTYPE val = f[idx(i, j, k, nx, ny)];
            rho += val;
            u_x += val * cx_const[k];
            u_y += val * cy_const[k];
        }
        if (rho > 1e-12f) {
            u_x /= rho;
            u_y /= rho;
        }
        DTYPE vel = sqrtf(u_x * u_x + u_y * u_y);
        velocity_mag[i + j * nx] = vel;
    }
}

//-----------------------------------------------------
// Kernel that copies the velocity magnitudes into RGBA
// for display, writing directly into a CUDA-mapped buffer
// (pbo) which has size (nx*ny*4 bytes).
// We'll do a grayscale: R=G=B=255*vel/U, A=255.
//-----------------------------------------------------
//__global__ void fill_pbo_kernel(unsigned char* pbo_ptr,
//    const DTYPE* velocity_mag,
//    int nx, int ny,
//    float U)
//{
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    int j = blockIdx.y * blockDim.y + threadIdx.y;
//    if (i < nx && j < ny) {
//        int idx_out = 4 * (i + j * nx); // RGBA
//        float v = velocity_mag[i + j * nx];
//
//        // Optionally clamp or scale
//        // float val = fminf(v / clampVal, 1.0f);  // scale velocity up to "clampVal"
//        unsigned char c = (unsigned char)(v / U * 255.0f);
//
//        pbo_ptr[idx_out + 0] = c;  // R
//        pbo_ptr[idx_out + 1] = c;  // G
//        pbo_ptr[idx_out + 2] = c;  // B
//        pbo_ptr[idx_out + 3] = 255;// A
//    }
//}

// Blue to red color map
__global__ void fill_pbo_kernel(unsigned char* pbo_ptr,
    const DTYPE* velocity_mag,
    int nx, int ny,
    float U)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        int idx_out = 4 * (i + j * nx); // RGBA
        float v = velocity_mag[i + j * nx];

        // Normalize velocity to [0, 1] based on U
        float t = fminf(v / U, 1.0f); // Clamp to [0, 1]

        // Linear interpolation from blue (0, 0, 255) to red (255, 0, 0)
        //unsigned char r = (unsigned char)(t * 255.0f);          // Red increases
        //unsigned char g = 0;                                    // Green stays 0
        //unsigned char b = (unsigned char)((1.0f - t) * 255.0f); // Blue decreases

        // Jet color bar (Blue to yellow to red color map)
        unsigned char r, g, b;
        if (t < 0.5f) {
            // Blue to yellow (0,0,255) -> (255,255,0)
            float s = t * 2.0f; // Map [0, 0.5] to [0, 1]
            r = (unsigned char)(s * 255.0f);         // 0 to 255
            g = (unsigned char)(s * 255.0f);         // 0 to 255
            b = (unsigned char)((1.0f - s) * 255.0f); // 255 to 0
        }
        else {
            // Yellow to red (255,255,0) -> (255,0,0)
            float s = (t - 0.5f) * 2.0f; // Map [0.5, 1] to [0, 1]
            r = 255;                                 // Stays 255
            g = (unsigned char)((1.0f - s) * 255.0f); // 255 to 0
            b = 0;                                   // Stays 0
        }

        pbo_ptr[idx_out + 0] = r;   // R
        pbo_ptr[idx_out + 1] = g;   // G
        pbo_ptr[idx_out + 2] = b;   // B
        pbo_ptr[idx_out + 3] = 255; // A (fully opaque)
    }
}

//-----------------------------------------------------
// Host routines for initialization and simulation
//-----------------------------------------------------
void initialize_simulation() {
    // Allocate device memory
    hipMalloc(&d_f, simSize);
    hipMalloc(&d_f_new, simSize);
    hipMalloc(&d_mask, maskSize);

    // Initialize f on host (uniform density=1, velocity=0)
    DTYPE* h_f = new DTYPE[nx * ny * numDirs];
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            DTYPE usq = 0.0f;
            for (int k = 0; k < numDirs; k++) {
                DTYPE cu = 3.0f * (cx[k] * 0.0f + cy[k] * 0.0f);
                h_f[idx_h(i, j, k, nx, ny)] = w[k] * 1.0f * (1.0f + cu + 0.5f * cu * cu - 1.5f * usq);
            }
        }
    }
    hipMemcpy(d_f, h_f, simSize, hipMemcpyHostToDevice);
    delete[] h_f;

    // Initialize mask (solid boundary)
    char* h_mask = new char[nx * ny];
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            if (i == 1 || i == nx - 2 || j == 1)
                h_mask[i + j * nx] = 1;
            else
                h_mask[i + j * nx] = 0;
        }
    }
    hipMemcpy(d_mask, h_mask, maskSize, hipMemcpyHostToDevice);
    delete[] h_mask;
}

// Runs one simulation step
void simulation_step() {
    dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    collision_kernel <<<gridDim,blockDim>>> (d_f, omega, nx, ny);
    hipDeviceSynchronize();

    streaming_kernel <<<gridDim,blockDim>>> (d_f, d_f_new, nx, ny);
    hipDeviceSynchronize();

    bounce_back_kernel <<<gridDim,blockDim>>> (d_f_new, d_mask, nx, ny);
    hipDeviceSynchronize();

    // Apply moving lid
    dim3 blockDim1(256);
    dim3 gridDim1((nx + blockDim1.x - 1) / blockDim1.x);
    moving_lid_kernel <<<gridDim1,blockDim1>>> (d_f_new, nx, ny, U);
    hipDeviceSynchronize();

    // Swap pointers
    DTYPE* temp = d_f;
    d_f = d_f_new;
    d_f_new = temp;
}

//-----------------------------------------------------
// Global (static) OpenGL/CUDA variables
//-----------------------------------------------------
static GLuint pbo = 0;                             // OpenGL pixel buffer object
static struct hipGraphicsResource* cuda_pbo = nullptr;
static DTYPE* d_velocity = nullptr;                // device array for velocity magnitude
static const int WIN_WIDTH = nx;                  // match your lattice dims
static const int WIN_HEIGHT = ny;
static int stepsPerFrame = 40;                  // how many LBM steps per OpenGL frame?

//-----------------------------------------------------
// Create the PBO and register it with CUDA
//-----------------------------------------------------
void create_pbo() {
    // Generate a buffer ID for the PBO
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    // Allocate the buffer (RGBA, 1 byte each, total 4 bytes/pixel)
    glBufferData(GL_PIXEL_UNPACK_BUFFER, WIN_WIDTH * WIN_HEIGHT * 4, NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(&cuda_pbo, pbo, hipGraphicsRegisterFlagsWriteDiscard);

    // Also allocate d_velocity
    hipMalloc((void**)&d_velocity, nx * ny * sizeof(DTYPE));
}

//-----------------------------------------------------
// Cleanup
//-----------------------------------------------------
void cleanup() {
    if (cuda_pbo) {
        hipGraphicsUnregisterResource(cuda_pbo);
        cuda_pbo = nullptr;
    }
    if (pbo) {
        glDeleteBuffers(1, &pbo);
        pbo = 0;
    }
    if (d_velocity) {
        hipFree(d_velocity);
        d_velocity = nullptr;
    }
    // Free LBM arrays
    if (d_f)       hipFree(d_f);
    if (d_f_new)   hipFree(d_f_new);
    if (d_mask)    hipFree(d_mask);
}

//-----------------------------------------------------
// Render callback: called by GLUT whenever we want to
// redraw the screen.
//-----------------------------------------------------
void display() {
    // 0) Record start time
    if (currentStep == 0) {
        startTime = std::chrono::high_resolution_clock::now();
    }

    // 1) Run LBM time steps
    for (int s = 0; s < stepsPerFrame; s++) {
        simulation_step();
		currentStep++;
    }

    // 2) Compute velocity magnitude on GPU
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    compute_velocity_field_kernel << <grid, block >> > (d_f, d_velocity, nx, ny);
    hipDeviceSynchronize();

    // 3) Map the PBO so we can write into it from CUDA
    hipGraphicsMapResources(1, &cuda_pbo, 0);
    unsigned char* d_pbo_ptr = nullptr;
    size_t num_bytes = 0;
    hipGraphicsResourceGetMappedPointer((void**)&d_pbo_ptr, &num_bytes, cuda_pbo);

    // 4) Fill PBO with color from velocity
    fill_pbo_kernel << <grid, block >> > (d_pbo_ptr, d_velocity, nx, ny, U);
    hipDeviceSynchronize();

    // 5) Unmap
    hipGraphicsUnmapResources(1, &cuda_pbo, 0);

    // 6) Clear screen and draw the pixel buffer
    glClear(GL_COLOR_BUFFER_BIT);
    glRasterPos2f(-1, -1); // draw from bottom-left
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    // The data we have is 8-bit RGBA
    glDrawPixels(WIN_WIDTH, WIN_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // 8) Compute and display time step, update per second, and MLUPS
    auto currentTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsedTime = currentTime - startTime;
    updatesPerSecond = float(currentStep) / elapsedTime.count();
    float framesPerSecond = updatesPerSecond / float(stepsPerFrame);
    mlups = (float(currentStep) * float(nx) * float(ny)) / (elapsedTime.count() * float(1e6));

    std::ostringstream oss;
    oss << "Time Step: " << currentStep 
        << "  UPS: " << std::fixed << std::setprecision(1) << updatesPerSecond 
        << "  MLUPS: " << std::fixed << std::setprecision(2) << mlups
        << "  FPS: " << std::fixed << std::setprecision(1) << framesPerSecond;
    std::string info = oss.str();

	glColor3f(1.0f, 1.0f, 1.0f); // white text
	glRasterPos2f(-0.95f, 0.95f); // upper-left corner
    for (char c : info) {
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, c);
    }

    // 9) Swap buffers
    glutSwapBuffers();
}

//-----------------------------------------------------
// Optional: Idle callback �C just request a new display.
// Could also use glutTimerFunc for fixed framerate.
//-----------------------------------------------------
void idle() {
    glutPostRedisplay();
}

//-----------------------------------------------------
// OpenGL init
//-----------------------------------------------------
void initGL(int* argc, char** argv) {
    // Initialize freeGLUT
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(WIN_WIDTH, WIN_HEIGHT);
    glutCreateWindow("LBM + OpenGL Visualization");

    printf("OpenGL version: %s\n", glGetString(GL_VERSION));
    printf("OpenGL vendor: %s\n", glGetString(GL_VENDOR));
    printf("OpenGL renderer: %s\n", glGetString(GL_RENDERER));

    // (Optional) Init GLEW
    GLenum err = glewInit();
    if (GLEW_OK != err) {
        fprintf(stderr, "Error initializing GLEW: %s\n", glewGetErrorString(err));
        exit(1);
    }

    // Create the PBO
    create_pbo();

    // Set callbacks
    glutDisplayFunc(display);
    glutIdleFunc(idle);

    // Basic GL state
    glDisable(GL_DEPTH_TEST);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    // Enable blend to display text
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
}

//-----------------------------------------------------
// Main
//-----------------------------------------------------
int main(int argc, char** argv) {
    // 1) Compute relaxation parameter
    nu = 3.0f * (U * float(nx) / Re) + 0.5f;
    omega = 1.0f / nu;
	// Print nu and omega
	printf("Relaxation time = %f, Omega = %f\n", nu, omega);
	// Print U and Re
	printf("U = %f, Re = %f\n", U, Re);

    // 2) Init the LBM arrays on GPU
    initialize_simulation();

    // 3) Init OpenGL and enter GLUT main loop
    initGL(&argc, argv);
    glutMainLoop();

    // 4) Cleanup (won't usually reach here unless you close the window)
    cleanup();
    return 0;
}
