#include "hip/hip_runtime.h"
﻿#include <chrono>
#include <iomanip>
#include <string>
#include <sstream>

// ====== CUDA libraries ======
#include "hip/hip_runtime.h"
#include ""

// ====== OpenGL / CUDA-OpenGL Interop ======
#include <GL/glew.h>       // if using GLEW
#include <GL/freeglut.h>   // if using freeGLUT
#include <cuda_gl_interop.h>

// Time info
int currentStep = 0;
std::chrono::time_point<std::chrono::high_resolution_clock> startTime;
float updatesPerSecond = 0.0f;
float mlups = 0.0f;
float framesPerSecond = 0.0f;

// Force the use of the NVIDIA GPU
#ifdef _WIN32
extern "C" {
    __declspec(dllexport) DWORD NvOptimusEnablement = 0x00000001;
}
#endif

//-----------------------------------------------------
// Lattice parameters and simulation constants
//-----------------------------------------------------
const int nx = 1024;
const int ny = 1024;
const int numDirs = 9;
typedef float DTYPE;

__constant__ int cx_const[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
__constant__ int cy_const[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
__constant__ DTYPE w_const[9] = {
    4.0 / 9.0,
    1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0,
    1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0
};

const int cx[9] = { 0,  1,  0, -1,  0,  1, -1, -1,  1 };
const int cy[9] = { 0,  0,  1,  0, -1,  1,  1, -1, -1 };
const DTYPE w[9] = {
    4.0 / 9.0,
    1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0,
    1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0, 1.0 / 36.0
};

DTYPE U = 0.3;  // Theoretically should be less than 0.577 (1/sqrt(3))
DTYPE Re = 35000.0;
DTYPE nu, tao, omega;  // nu = U*nx/Re; tao = 3*nu+0.5; omega = 1/tao

// Simulation arrays (device pointers)
DTYPE* d_f = nullptr, * d_f_new = nullptr;
char* d_mask = nullptr;

size_t simSize = nx * ny * numDirs * sizeof(DTYPE);
size_t maskSize = nx * ny * sizeof(char);

//-----------------------------------------------------
// Helper device and host inline for indexing
//-----------------------------------------------------
__device__ inline int idx(int i, int j, int k, int nx, int ny) {
    return i + j * nx + k * nx * ny;
}
inline int idx_h(int i, int j, int k, int nx, int ny) {
    return i + j * nx + k * nx * ny;
}

//-----------------------------------------------------
// CUDA kernels for the LBM solver
//-----------------------------------------------------
__global__ void collision_kernel(DTYPE* f, DTYPE omega, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        DTYPE rho = 0.0;
        DTYPE u_x = 0.0;
        DTYPE u_y = 0.0;
        for (int k = 0; k < numDirs; k++) {
            DTYPE val = f[idx(i, j, k, nx, ny)];
            rho += val;
            u_x += val * cx_const[k];
            u_y += val * cy_const[k];
        }
        if (rho > DTYPE(0.0)) {
            u_x /= rho;
            u_y /= rho;
        }
        DTYPE usqr = u_x * u_x + u_y * u_y;
        for (int k = 0; k < numDirs; k++) {
            DTYPE cu = DTYPE(3.0) * (cx_const[k] * u_x + cy_const[k] * u_y);
            DTYPE feq = w_const[k] * rho * (DTYPE(1.0) + cu + DTYPE(0.5) * cu * cu - DTYPE(1.5) * usqr);
            f[idx(i, j, k, nx, ny)] = (DTYPE(1.0) - omega) * f[idx(i, j, k, nx, ny)] + omega * feq;
        }
    }
}

__global__ void streaming_kernel(DTYPE* f_in, DTYPE* f_out, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
            for (int k = 0; k < numDirs; k++) {
                int ip = i - cx_const[k];
                int jp = j - cy_const[k];
                f_out[idx(i, j, k, nx, ny)] = f_in[idx(ip, jp, k, nx, ny)];
            }
        }
        else {
            // For simplicity, no wrap-around. Just copy as-is at borders
            for (int k = 0; k < numDirs; k++) {
                f_out[idx(i, j, k, nx, ny)] = f_in[idx(i, j, k, nx, ny)];
            }
        }
    }
}

__global__ void bounce_back_kernel(DTYPE* f, char* mask, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        if (mask[i + j * nx] == 1) {
            // Swap east (1) and west (3)
            int idx1 = idx(i, j, 1, nx, ny);
            int idx3 = idx(i, j, 3, nx, ny);
            DTYPE tmp = f[idx1];
            f[idx1] = f[idx3];
            f[idx3] = tmp;
            // Swap north (2) and south (4)
            int idx2 = idx(i, j, 2, nx, ny);
            int idx4 = idx(i, j, 4, nx, ny);
            tmp = f[idx2];
            f[idx2] = f[idx4];
            f[idx4] = tmp;
            // Swap NE (5) and SW (7)
            int idx5 = idx(i, j, 5, nx, ny);
            int idx7 = idx(i, j, 7, nx, ny);
            tmp = f[idx5];
            f[idx5] = f[idx7];
            f[idx7] = tmp;
            // Swap NW (6) and SE (8)
            int idx6 = idx(i, j, 6, nx, ny);
            int idx8 = idx(i, j, 8, nx, ny);
            tmp = f[idx6];
            f[idx6] = f[idx8];
            f[idx8] = tmp;
        }
    }
}

__global__ void moving_lid_kernel(DTYPE* f, int nx, int ny, DTYPE U) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nx) {
        int j = ny - 2;
        // Zou-He velocity BC
        //DTYPE rho = f[idx(i, j, 0, nx, ny)] + f[idx(i, j, 1, nx, ny)] + f[idx(i, j, 3, nx, ny)]
        //    + 2.0f * (f[idx(i, j, 2, nx, ny)] + f[idx(i, j, 5, nx, ny)] + f[idx(i, j, 6, nx, ny)]);
        //    f[idx(i, j, 4, nx, ny)] = f[idx(i, j, 2, nx, ny)];
        //    f[idx(i, j, 7, nx, ny)] = f[idx(i, j, 5, nx, ny)]
        //        + 0.5f * (f[idx(i, j, 1, nx, ny)] - f[idx(i, j, 3, nx, ny)])
        //            - 0.5f * rho * U;
        //        f[idx(i, j, 8, nx, ny)] = f[idx(i, j, 6, nx, ny)]
        //            - 0.5f * (f[idx(i, j, 1, nx, ny)] - f[idx(i, j, 3, nx, ny)])
        //                + 0.5f * rho * U;

		// Mid-grid velocity BC
		f[idx(i, j, 4, nx, ny)] = f[idx(i, j, 2, nx, ny)];
		f[idx(i, j, 7, nx, ny)] = f[idx(i, j, 5, nx, ny)] - DTYPE(1.0) / DTYPE(6.0) * U;
		f[idx(i, j, 8, nx, ny)] = f[idx(i, j, 6, nx, ny)] + DTYPE(1.0) / DTYPE(6.0) * U;
    }
}

// Helper device function for sqrt
template<typename T>
__device__ __forceinline__ T device_sqrt(T x) {
    if constexpr (std::is_same<T, float>::value) {
        return sqrtf(x);
    }
    else {
        return sqrt(x);
    }
}

//-----------------------------------------------------
// Kernel to compute velocity magnitude into a float array
//-----------------------------------------------------
__global__ void compute_velocity_field_kernel(const DTYPE* f, DTYPE* velocity_mag, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        DTYPE rho = 0.0;
        DTYPE u_x = 0.0;
        DTYPE u_y = 0.0;
        for (int k = 0; k < numDirs; k++) {
            DTYPE val = f[idx(i, j, k, nx, ny)];
            rho += val;
            u_x += val * cx_const[k];
            u_y += val * cy_const[k];
        }
        if (rho > DTYPE(1e-12)) {
            u_x /= rho;
            u_y /= rho;
        }
        DTYPE vel = device_sqrt(u_x * u_x + u_y * u_y);
        velocity_mag[i + j * nx] = vel;
    }
}

//-----------------------------------------------------
// Kernel that copies the velocity magnitudes into RGBA
// for display, writing directly into a CUDA-mapped buffer
// (pbo) which has size (nx*ny*4 bytes).

// Blue to red color map
__global__ void fill_pbo_kernel(unsigned char* pbo_ptr,
    const DTYPE* velocity_mag,
    int nx, int ny,
    float U)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < nx && j < ny) {
        int idx_out = 4 * (i + j * nx); // RGBA
        float v = velocity_mag[i + j * nx];

        // Normalize velocity to [0, 1] based on U
        float t = fminf(v / float(U), 1.0f); // Clamp to [0, 1]

        // Jet color bar (Blue to yellow to red color map)
        unsigned char r, g, b;
        if (t < 0.5f) {
            // Blue to yellow (0,0,255) -> (255,255,0)
            float s = t * 2.0f; // Map [0, 0.5] to [0, 1]
            r = (unsigned char)(s * 255.0f);         // 0 to 255
            g = (unsigned char)(s * 255.0f);         // 0 to 255
            b = (unsigned char)((1.0f - s) * 255.0f); // 255 to 0
        }
        else {
            // Yellow to red (255,255,0) -> (255,0,0)
            float s = (t - 0.5f) * 2.0f; // Map [0.5, 1] to [0, 1]
            r = 255;                                 // Stays 255
            g = (unsigned char)((1.0f - s) * 255.0f); // 255 to 0
            b = 0;                                   // Stays 0
        }

        pbo_ptr[idx_out + 0] = r;   // R
        pbo_ptr[idx_out + 1] = g;   // G
        pbo_ptr[idx_out + 2] = b;   // B
        pbo_ptr[idx_out + 3] = 255; // A (fully opaque)
    }
}

//-----------------------------------------------------
// Host routines for initialization and simulation
//-----------------------------------------------------
void initialize_simulation(DTYPE rho0, DTYPE ux0, DTYPE uy0) {
    // Allocate device memory
    hipMalloc(&d_f, simSize);
    hipMalloc(&d_f_new, simSize);
    hipMalloc(&d_mask, maskSize);

    // Initialize f on host (uniform density=1, velocity=0)
    DTYPE* h_f = new DTYPE[nx * ny * numDirs];
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            DTYPE usq = ux0 * ux0 + uy0 * uy0;
            for (int k = 0; k < numDirs; k++) {
                DTYPE cu = DTYPE(3.0) * (cx[k] * ux0 + cy[k] * uy0);
                h_f[idx_h(i, j, k, nx, ny)] = w[k] * rho0 * (DTYPE(1.0) + cu + DTYPE(0.5) * cu * cu - DTYPE(1.5) * usq);
            }
        }
    }
    hipMemcpy(d_f, h_f, simSize, hipMemcpyHostToDevice);
    delete[] h_f;

    // Initialize mask (solid boundary)
    char* h_mask = new char[nx * ny];
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            if (i == 1 || i == nx - 2 || j == 1)
                h_mask[i + j * nx] = 1;
            else
                h_mask[i + j * nx] = 0;
        }
    }
    hipMemcpy(d_mask, h_mask, maskSize, hipMemcpyHostToDevice);
    delete[] h_mask;
}

//--------------------------------------------------------------------------------------------
// Runs one simulation step
// Since all kernel launches use the default stream, removing all the hipDeviceSynchronize()
// calls will not affect the correctness of the LBM calculation, but increase the performance
//--------------------------------------------------------------------------------------------
void simulation_step() {
    dim3 blockDim(16, 16);
    dim3 gridDim((nx + blockDim.x - 1) / blockDim.x, (ny + blockDim.y - 1) / blockDim.y);

    collision_kernel <<<gridDim, blockDim>>> (d_f, omega, nx, ny);
    //hipDeviceSynchronize();

    streaming_kernel <<<gridDim, blockDim>>> (d_f, d_f_new, nx, ny);
    //hipDeviceSynchronize();

    bounce_back_kernel <<<gridDim, blockDim>>> (d_f_new, d_mask, nx, ny);
    //hipDeviceSynchronize();

    // Apply moving lid
    dim3 blockDim1(256);
    dim3 gridDim1((nx + blockDim1.x - 1) / blockDim1.x);
    moving_lid_kernel <<<gridDim1, blockDim1>>> (d_f_new, nx, ny, U);
    //hipDeviceSynchronize();

    // Swap pointers
    DTYPE* temp = d_f;
    d_f = d_f_new;
    d_f_new = temp;
}

//-----------------------------------------------------
// Global (static) OpenGL/CUDA variables
//-----------------------------------------------------
static GLuint pbo = 0;                             // OpenGL pixel buffer object
static struct hipGraphicsResource* cuda_pbo = nullptr;
static DTYPE* d_velocity = nullptr;                // device array for velocity magnitude
static const int WIN_WIDTH = nx;                  // match your lattice dims
static const int WIN_HEIGHT = ny;
static int stepsPerFrame = 100;                  // how many LBM steps per OpenGL frame?

//-----------------------------------------------------
// Create the PBO and register it with CUDA
//-----------------------------------------------------
void create_pbo() {
    // Generate a buffer ID for the PBO
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    // Allocate the buffer (RGBA, 1 byte each, total 4 bytes/pixel)
    glBufferData(GL_PIXEL_UNPACK_BUFFER, WIN_WIDTH * WIN_HEIGHT * 4, NULL, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // Register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(&cuda_pbo, pbo, hipGraphicsRegisterFlagsWriteDiscard);

    // Also allocate d_velocity
    hipMalloc((void**)&d_velocity, nx * ny * sizeof(DTYPE));
}

//-----------------------------------------------------
// Cleanup
//-----------------------------------------------------
void cleanup() {
    if (cuda_pbo) {
        hipGraphicsUnregisterResource(cuda_pbo);
        cuda_pbo = nullptr;
    }
    if (pbo) {
        glDeleteBuffers(1, &pbo);
        pbo = 0;
    }
    if (d_velocity) {
        hipFree(d_velocity);
        d_velocity = nullptr;
    }
    // Free LBM arrays
    if (d_f)       hipFree(d_f);
    if (d_f_new)   hipFree(d_f_new);
    if (d_mask)    hipFree(d_mask);
}

//-----------------------------------------------------
// Render callback: called by GLUT whenever we want to
// redraw the screen.
//-----------------------------------------------------
void display() {
    // 0) Record start time
    if (currentStep == 0) {
        startTime = std::chrono::high_resolution_clock::now();
    }

    // 1) Run LBM time steps
    for (int s = 0; s < stepsPerFrame; s++) {
        simulation_step();
		currentStep++;
    }

    // 2) Compute velocity magnitude on GPU
    dim3 block(16, 16);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    compute_velocity_field_kernel <<<grid, block>>> (d_f, d_velocity, nx, ny);
    hipDeviceSynchronize();

    // 3) Map the PBO so we can write into it from CUDA
    hipGraphicsMapResources(1, &cuda_pbo, 0);
    unsigned char* d_pbo_ptr = nullptr;
    size_t num_bytes = 0;
    hipGraphicsResourceGetMappedPointer((void**)&d_pbo_ptr, &num_bytes, cuda_pbo);

    // 4) Fill PBO with color from velocity
    fill_pbo_kernel <<<grid, block>>> (d_pbo_ptr, d_velocity, nx, ny, U);
    hipDeviceSynchronize();

    // 5) Unmap
    hipGraphicsUnmapResources(1, &cuda_pbo, 0);

    // 6) Clear screen and draw the pixel buffer
    glClear(GL_COLOR_BUFFER_BIT);
    glRasterPos2f(-1, -1); // draw from bottom-left
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    // The data we have is 8-bit RGBA
    glDrawPixels(WIN_WIDTH, WIN_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, 0);

    // 7) Compute and display time step, update per second, and MLUPS
    auto currentTime = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsedTime = currentTime - startTime;
    updatesPerSecond = float(currentStep) / elapsedTime.count();
    framesPerSecond = updatesPerSecond / float(stepsPerFrame);
    mlups = (float(currentStep) * float(nx) * float(ny)) / (elapsedTime.count() * float(1e6));

    std::ostringstream oss;
    oss << "Time Step: " << currentStep 
        << "  UPS: " << std::fixed << std::setprecision(1) << updatesPerSecond 
        << "  MLUPS: " << std::fixed << std::setprecision(2) << mlups
        << "  FPS: " << std::fixed << std::setprecision(1) << framesPerSecond;
    std::string info = oss.str();

	glColor3f(1.0f, 1.0f, 1.0f); // white text
	glRasterPos2f(-0.95f, 0.95f); // upper-left corner
    for (char c : info) {
        glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18, c);
    }

    // 8) Swap buffers
    glutSwapBuffers();
}

//-----------------------------------------------------
// Optional: Idle callback �C just request a new display.
// Could also use glutTimerFunc for fixed framerate.
//-----------------------------------------------------
void idle() {
    glutPostRedisplay();
}

//-----------------------------------------------------
// OpenGL init
//-----------------------------------------------------
void initGL(int* argc, char** argv) {
    // Initialize freeGLUT
    glutInit(argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
    glutInitWindowSize(WIN_WIDTH, WIN_HEIGHT);
    glutCreateWindow("LBM + OpenGL Visualization");

    printf("OpenGL version: %s\n", glGetString(GL_VERSION));
    printf("OpenGL vendor: %s\n", glGetString(GL_VENDOR));
    printf("OpenGL renderer: %s\n", glGetString(GL_RENDERER));

    // Initialize GLEW
    GLenum err = glewInit();
    if (GLEW_OK != err) {
        fprintf(stderr, "Error initializing GLEW: %s\n", glewGetErrorString(err));
        exit(1);
    }

    // Create the PBO
    create_pbo();

    // Set callbacks
    glutDisplayFunc(display);
    glutIdleFunc(idle);

    // Basic GL state
    glDisable(GL_DEPTH_TEST);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);

    // Enable blend to display text
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);
}

//-----------------------------------------------------
// Main
//-----------------------------------------------------
int main(int argc, char** argv) {
    // 1) Compute relaxation parameter
	nu = U * DTYPE(nx) / Re;
    tao = DTYPE(3.0) * nu + DTYPE(0.5);
    omega = DTYPE(1.0) / tao;
	DTYPE rho0 = 1.0;
    DTYPE ux0 = 0.0;
	DTYPE uy0 = 0.0;
	// Print nu and tao and omega
	printf("Viscosity = %f, Relaxation time = %f, Omega = %f\n", nu, tao, omega);
	// Print U and Re
	printf("U = %f, Re = %f\n", U, Re);

    // 2) Init the LBM arrays on GPU
    initialize_simulation(rho0, ux0, uy0);

    // 3) Init OpenGL and enter GLUT main loop
    initGL(&argc, argv);
    glutMainLoop();

    // 4) Cleanup (won't usually reach here unless you close the window)
    cleanup();
    return 0;
}
